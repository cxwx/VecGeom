#include "hip/hip_runtime.h"
/// \file Raytracer.cu
/// \author Guilherme Amadio. Rewritten to use navigation from common code by Andrei Gheata. 

#include "VecGeom/benchmarking/Raytracer.h"

#include <VecGeom/base/Transformation3D.h>
#include <VecGeom/management/GeoManager.h>
#include <VecGeom/management/CudaManager.h>
#include <VecGeom/navigation/NavigationState.h>
#include <VecGeom/volumes/PlacedVolume.h>
#include <VecGeom/benchmarking/Raytracer.h>

#include <cassert>
#include <cstdio>

using namespace vecgeom;

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
  if (result) {
    fprintf(stderr, "CUDA error = %s at %s:%d\n", hipGetErrorString(result), file, line);
    hipDeviceReset();
    exit(1);
  }
}

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

__global__
void RenderKernel(RaytracerData_t rtdata, char *input_buffer, unsigned char *output_buffer)
{
  int px = threadIdx.x + blockIdx.x * blockDim.x;
  int py = threadIdx.y + blockIdx.y * blockDim.y;

  if ((px >= rtdata.fSize_px) || (py >= rtdata.fSize_py)) return;

  int pixel_index = 4 * (py * rtdata.fSize_px + px);

  Color_t pixel_color = Raytracer::RaytraceOne(px, py, rtdata, input_buffer);

  output_buffer[pixel_index + 0] = pixel_color.fComp.red;
  output_buffer[pixel_index + 1] = pixel_color.fComp.green;
  output_buffer[pixel_index + 2] = pixel_color.fComp.blue;
  output_buffer[pixel_index + 3] = 255;
}

int RaytraceBenchmarkGPU(vecgeom::cuda::VPlacedVolume const* const world, int px, int py, 
                         int model, int view, double zoom,
			 double screenx, double screeny, double screenz,
                         double upx, double upy, double upz,
                         int lightcol, int objcol, int maxdepth, int vdepth)
{
  using Vector3 = cuda::Vector3D<double>;
  RaytracerData_t rtdata;
  
  rtdata.fScreenPos.Set(screenx, screeny, screenz);
  rtdata.fUp.Set(upx, upy, upz);
  rtdata.fZoom       = zoom;
  rtdata.fModel      = (ERTmodel)model;
  rtdata.fView       = (ERTView)view;
  rtdata.fSize_px    = px;
  rtdata.fSize_py    = py;
  rtdata.fLightColor = lightcol;
  rtdata.fObjColor   = objcol;
  rtdata.fVisDepth   = vdepth;
  rtdata.fMaxDepth   = maxdepth;

  Raytracer::InitializeModel(world, rtdata);
  rtdata.Print();

  // Allocate ray data and output data on the device
  size_t statesize = NavigationState::SizeOfInstance(rtdata.fMaxDepth);
  size_t raysize = Ray_t::SizeOfInstance(rtdata.fMaxDepth);

  printf("=== Allocating %.3f MB of ray data on the device\n", (float)rtdata.fNrays * raysize / 1048576);
  //char *input_buffer_gpu = nullptr;
  char *input_buffer = new char[statesize + rtdata.fNrays * raysize];
  checkCudaErrors(hipMallocManaged((void **)&input_buffer, statesize + rtdata.fNrays * raysize));

  unsigned char *output_buffer = nullptr;
  checkCudaErrors(hipMallocManaged((void **)&output_buffer, 4 * sizeof(unsigned char) * rtdata.fSize_px * rtdata.fSize_py));

  // Load and synchronize the geometry on the GPU
  vecgeom::cxx::CudaManager::Instance().LoadGeometry((vecgeom::cxx::VPlacedVolume*) world);
  vecgeom::cxx::CudaManager::Instance().Synchronize();

  // CudaManager is altering the stack size... setting an appropriate value
  size_t def_stack_limit = 0, def_heap_limit = 0;
  hipDeviceGetLimit( &def_stack_limit, hipLimitStackSize);
  hipDeviceGetLimit( &def_heap_limit, hipLimitMallocHeapSize);
  std::cout << "=== hipLimitStackSize = " << def_stack_limit << "  hipLimitMallocHeapSize = " << def_heap_limit << std::endl;
  auto err = hipDeviceSetLimit(hipLimitStackSize, 8192);
  hipDeviceGetLimit( &def_stack_limit, hipLimitStackSize);
  std::cout << "=== CUDA thread stack size limit set now to: " << def_stack_limit << std::endl;
  
  auto gpu_world = vecgeom::cxx::CudaManager::Instance().world_gpu();
  assert(gpu_world && "GPU world volume is a null pointer");

  // Initialize the navigation state for the view point
  auto vpstate = NavigationState::MakeInstanceAt(rtdata.fMaxDepth, (void *)(input_buffer));
  Raytracer::LocateGlobalPoint(rtdata.fWorld, rtdata.fStart, *vpstate, true);
  rtdata.fVPstate = vpstate;
  rtdata.fWorld   = gpu_world;

  rtdata.Print();

// Construct rays in place
  char *raybuff = input_buffer + statesize;
  for (int iray = 0; iray < rtdata.fNrays; ++iray)
    Ray_t::MakeInstanceAt(raybuff + iray * raysize, rtdata.fMaxDepth);

  dim3 blocks(rtdata.fSize_px / 8 + 1, rtdata.fSize_py / 8 + 1), threads(8, 8);
  RenderKernel<<<blocks, threads>>>(rtdata, input_buffer, output_buffer);

  checkCudaErrors(hipGetLastError());
  checkCudaErrors(hipDeviceSynchronize());

  write_ppm("output.ppm", output_buffer, rtdata.fSize_px, rtdata.fSize_py);

  checkCudaErrors(hipFree(input_buffer));
  checkCudaErrors(hipFree(output_buffer));
  return 0;
}

